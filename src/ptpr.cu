#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include <iostream>

#include "main.hpp"
#include "ptpr.cuh"

#if (__CUDACC_VER_MAJOR__ <= 11) && (__CUDACC_VER_MINOR__ < 4)
#pragma message("Using Experimental Features")
#define EXPERIMENTAL_ASYNC_COPY
#endif

namespace cg = cooperative_groups;
#ifdef EXPERIMENTAL_ASYNC_COPY
using namespace nvcuda::experimental;
#endif

using namespace PTPR_CU;

/**
 * for dynamic shared memory use
 */
extern __shared__ float array[];

/**
 * @brief      PTPR main kernel
 *
 * @param      a     { parameter_description }
 * @param      c     { parameter_description }
 * @param      rhs   The right hand side
 * @param      x     { parameter_description }
 * @param[in]  n     { parameter_description }
 * @param[in]  s     { parameter_description }
 */
__global__ void PTPR_CU::tpr_ker(float *a, float *c, float *rhs, float *x,
                                 int n, int s) {
    cg::grid_group tg = cg::this_grid();
    cg::thread_block tb = cg::this_thread_block();
    assert(tg.is_valid());
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int st = idx / s * s;
    int ed = st + s - 1;

    // local copy
    // sha[0:s], shc[0:s], shrhs[0:s]
    __shared__ float *sha, *shc, *shrhs;
    sha = (float *)array;
    shc = (float *)&array[s];
    shrhs = (float *)&array[2 * s];

    // make local copy on shared memory
#ifdef EXPERIMENTAL_ASYNC_COPY
    pipeline pipe;
    memcpy_async(sha[idx - st], a[idx], pipe);
    memcpy_async(shc[idx - st], c[idx], pipe);
    memcpy_async(shrhs[idx - st], rhs[idx], pipe);
#else
    cg::memcpy_async(tb, sha, &a[st], sizeof(float) * s);
    cg::memcpy_async(tb, shc, &c[st], sizeof(float) * s);
    cg::memcpy_async(tb, shrhs, &rhs[st], sizeof(float) * s);
#endif

    Equation eq;
    eq.a = sha;
    eq.c = shc;
    eq.rhs = shrhs;
    eq.x = x;

    TPR_Params params;
    params.n = n;
    params.s = s;
    params.idx = idx;
    params.st = st;
    params.ed = ed;

    float tmp_aa, tmp_cc, tmp_rr;
    // bkups, .x -> a, .y -> c, .z -> rhs
    float3 bkup_st, bkup_ed;

#ifdef EXPERIMENTAL_ASYNC_COPY
    pipe.commit_and_wait();
#else
    cg::wait(tb);
#endif
    tpr_st1_ker(tb, eq, params);

    tpr_inter(tb, eq, bkup_st, params);

    tb.sync();

    // copy back
    // since `tpr_inter_global` and stage 2 are global operations,
    // eq.* should hold address in global memory
    a[idx] = sha[idx - st];
    c[idx] = shc[idx - st];
    rhs[idx] = shrhs[idx - st];
    eq.a = a;
    eq.c = c;
    eq.rhs = rhs;

    tg.sync();

    tpr_inter_global(tb, eq, bkup_ed, params);

    // PCR
    for (int p = static_cast<int>(log2f(static_cast<double>(s))) + 1;
         p <= static_cast<int>(log2f(static_cast<double>(n))); p++) {
        if (idx < n && idx == ed) {
            // reduction
            int u = 1 << (p - 1);  // offset
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < 0) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = a[lidx];
                ckl = c[lidx];
                rkl = rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx >= n) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = a[ridx];
                ckr = c[ridx];
                rkr = rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * a[idx] - akr * c[idx]);

            tmp_aa = -inv_diag_k * akl * a[idx];
            tmp_cc = -inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);
        }

        tg.sync();

        if (idx < n && idx == ed) {
            // copy back
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;
        }

        tg.sync();
    }

    tpr_st2_copyback(tb, rhs, x, n, s);

    // stage 3
    // assert sh* has data
    if (idx < n && idx == st) {
        // idx - st == 0
        sha[idx - st] = bkup_st.x;
        shc[idx - st] = bkup_st.y;
        shrhs[idx - st] = bkup_st.z;
    }

    // should be same condition as tpr_inter_global
    if (idx < n && idx == ed) {
        sha[s - 1] = bkup_ed.x;
        shc[s - 1] = bkup_ed.y;
        shrhs[s - 1] = bkup_ed.z;
    }

    tg.sync();

    // tpr_st3_ker use shared memory
    eq.a = sha;
    eq.c = shc;
    eq.rhs = shrhs;
    tpr_st3_ker(tb, eq, params);

    return;
}

/**
 * @brief      PTPR Stage 1
 *
 * @param          tb      cg::thread_block
 * @param[in,out]  eq      Equation. `eq.a, eq.c, eq.rhs` should be address in
 * shared memory
 * @param[in]      params  The parameters of PTPR
 */
__device__ void PTPR_CU::tpr_st1_ker(cg::thread_block &tb, Equation eq,
                                     TPR_Params const &params) {
    int idx = params.idx;
    int i = tb.thread_index().x;
    int n = params.n, s = params.s;
    float tmp_aa, tmp_cc, tmp_rr;
    float *sha = eq.a, *shc = eq.c, *shrhs = eq.rhs;
    assert(__isShared((void *)sha));
    assert(__isShared((void *)shc));
    assert(__isShared((void *)shrhs));

    for (int p = 1; p <= static_cast<int>(log2f(static_cast<double>(s))); p++) {
        if (idx < n) {
            // reduction
            int u = 1 << (p - 1);  // offset
            int lidx = i - u;
            float akl, ckl, rkl;
            if (lidx < 0) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = sha[lidx];
                ckl = shc[lidx];
                rkl = shrhs[lidx];
            }
            int ridx = i + u;
            float akr, ckr, rkr;
            if (ridx >= s) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = sha[ridx];
                ckr = shc[ridx];
                rkr = shrhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * sha[i] - akr * shc[i]);

            tmp_aa = -inv_diag_k * akl * sha[i];
            tmp_cc = -inv_diag_k * ckr * shc[i];
            tmp_rr = inv_diag_k * (shrhs[i] - rkl * sha[i] - rkr * shc[i]);
        }

        tb.sync();

        if (idx < n) {
            // copy back
            sha[i] = tmp_aa;
            shc[i] = tmp_cc;
            shrhs[i] = tmp_rr;
        }

        tb.sync();
    }
}

/**
 * @brief      PTPR Intermediate stage 1
 *
 * Update E_{st} by E_{ed}
 *
 * @param          tb      cg::thread_block
 * @param[in,out]  eq      Equation. `eq.a, eq.c, eq.rhs` should be address in
 * shared memory
 * @param[out]     bkup    The bkup for stage 3 use. bkup->x: a, bkup->y: c,
 * bkup->z: rhs
 * @param[in]      params  The parameters of PTPR
 */
__device__ void PTPR_CU::tpr_inter(cg::thread_block &tb, Equation eq,
                                   float3 &bkup, TPR_Params const &params) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    float tmp_aa, tmp_cc, tmp_rr;

    if ((idx < params.n) && (idx == params.st)) {
        int k = idx - params.st;  // == 0,
        /**
        FIXME: writing 0 cause compile error
        nvcc: V11.4.48, cuda: 11.4
        ```
        Invalid bitcast
        float* bitcast ([0 x float] addrspace(3)* @array to float*)
        Error: Broken function found, compilation aborted!
        ```
        **/
        int kr = params.s - 1;
        float ak = eq.a[k], akr = eq.a[kr];
        float ck = eq.c[k], ckr = eq.c[kr];
        float rhsk = eq.rhs[k], rhskr = eq.rhs[kr];

        float inv_diag_k = 1.0 / (1.0 - akr * ck);

        tmp_aa = inv_diag_k * ak;
        tmp_cc = -inv_diag_k * ckr * ck;
        tmp_rr = inv_diag_k * (rhsk - rhskr * ck);

        // idx == st
        bkup.x = eq.a[k];
        bkup.y = eq.c[k];
        bkup.z = eq.rhs[k];

        eq.a[k] = tmp_aa;
        eq.c[k] = tmp_cc;
        eq.rhs[k] = tmp_rr;
    }
}

/**
 * @brief      PTPR Intermediate stage GLOBAL
 *
 * Update E_{st-1} by E_{st}
 *
 * @param          tb      cg::thread_block
 * @param[in,out]  eq      Equation. `eq.a, eq.c, eq.rhs` should be address in
 * GLOBAL memory
 * @param[out]     bkup    The bkup for stage 3 use. bkup->x: a, bkup->y: c,
 * bkup->z: rhs
 * @param[in]      params  The parameters of PTPR
 */
__device__ void PTPR_CU::tpr_inter_global(cg::thread_block &tb, Equation eq,
                                          float3 &bkup,
                                          TPR_Params const &params) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int ed = params.ed;

    if (idx < params.n - 1 && idx == ed) {
        int k = idx, kr = idx + 1;  // (k, kr) = (st-1, st)
        float ak = eq.a[k], akr = eq.a[kr];
        float ck = eq.c[k], ckr = eq.c[kr];
        float rhsk = eq.rhs[k], rhskr = eq.rhs[kr];
        float inv_diag_k = 1.0 / (1.0 - akr * ck);

        bkup.x = eq.a[idx];
        bkup.y = eq.c[idx];
        bkup.z = eq.rhs[idx];

        eq.a[k] = inv_diag_k * ak;
        eq.c[k] = -inv_diag_k * ckr * ck;
        eq.rhs[k] = inv_diag_k * (rhsk - rhskr * ck);
    } else if (idx == params.n - 1) {
        bkup.x = eq.a[idx];
        bkup.y = eq.c[idx];
        bkup.z = eq.rhs[idx];
    }
}

/**
 * @brief      copy the answer from stage 2 PCR
 *
 * @note assert { rhs[i] | i \in [0, n), i % (s-1) == 0 } has the answer
 *
 * @param        tb    cg::thread_block
 * @param[in]    rhs   The right hand side. Address in GLOBAL memory.
 * @param[out]   x     The answer array. Address in GLOBAL memory.
 * @param[in]    n     Parameter
 * @param[in]    s     Parameter
 */
__device__ void PTPR_CU::tpr_st2_copyback(cg::thread_block &tb, float *rhs,
                                          float *x, int n, int s) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int st = idx / s * s;
    int ed = st + s - 1;

    if (idx < n && idx == ed) {
        x[idx] = rhs[idx];
    }
}

/**
 * @brief      PTPR Stage 3
 *
 * @param          tb      cg::thread_block
 * @param[in,out]  eq      Equation. `eq.a, eq.c, eq.rhs` should be address in
 * shared memory
 * @param[in]      params  The parameters of PTPR
 */
__device__ void PTPR_CU::tpr_st3_ker(cg::thread_block &tb, Equation eq,
                                     TPR_Params const &params) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int i = tb.thread_index().x;
    int st = params.st;
    int ed = params.ed;
    int n = params.n, s = params.s;
    assert(__isShared((void *)eq.a));
    assert(__isShared((void *)eq.c));
    assert(__isShared((void *)eq.rhs));
    assert(__isGlobal((void *)eq.x));

    if (idx < n) {
        int lidx = max(0, st - 1);

        float key = 1.0 / eq.c[s - 1] *
                    (eq.rhs[s - 1] - eq.a[s - 1] * eq.x[lidx] - eq.x[ed]);
        if (eq.c[s - 1] == 0.0) {
            key = 0.0;
        }

        eq.x[idx] = eq.rhs[i] - eq.a[i] * eq.x[lidx] - eq.c[i] * key;
    }
    return;
}

__global__ void PTPR_CU::pcr_ker(float *a, float *c, float *rhs, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp_aa, tmp_cc, tmp_rr;

    if (idx < n) {
        for (int p = 1; p <= static_cast<int>(log2f(static_cast<double>(n)));
             p++) {
            // reduction
            int u = 1 << (p - 1);  // offset
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < 0) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = a[lidx];
                ckl = c[lidx];
                rkl = rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx >= n) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = a[ridx];
                ckr = c[ridx];
                rkr = rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * a[idx] - akr * c[idx]);

            tmp_aa = -inv_diag_k * akl * a[idx];
            tmp_cc = -inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);

            __syncthreads();

            // copy back
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;

            __syncthreads();
        }
    }
}

#define CU_CHECK(expr)                                                     \
    {                                                                      \
        hipError_t t = expr;                                              \
        if (t != hipSuccess) {                                            \
            fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", \
                    hipGetErrorString(t), t, __FILE__, __LINE__);         \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

/**
 * @brief      Helper function for ptpr_cu
 *
 * 1. check if device support cooperative launch
 * 2. allocate device memory for compute
 * 3. launch kernel `PTPR_CU::tpr_cu`
 * 4. copy the answer from device to host
 * 5. free device memory
 *
 * @param[in]  a     { parameter_description }
 * @param[in]  c     { parameter_description }
 * @param[in]  rhs   The right hand side
 * @param[out] x     x[0:n] for the answer
 * @param[in]  n     { parameter_description }
 * @param[in]  s     { parameter_description }
 */
void PTPR_CU::ptpr_cu(float *a, float *c, float *rhs, float *x, int n, int s) {
    int dev = 0;
    int size = n * sizeof(float);

    // Device
    float *d_a, *d_c, *d_r;  // device copies of a, c, rhs
    float *d_x;
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));
    CU_CHECK(hipMalloc((void **)&d_x, size));

    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    // launch configuration
    void *kernel_args[] = {&d_a, &d_c, &d_r, &d_x, &n, &s};
    auto config = tpr_launch_config(n, s, dev);
    // auto [dim_grid, dim_block, shmem_size] = rhs; not supported
    auto dim_grid = std::get<0>(config);
    auto dim_block = std::get<1>(config);
    auto shmem_size = std::get<2>(config);

#ifdef TPR_PERF
    {
        time_ms elapsed = 0;
        pmcpp::DeviceTimer timer;
        timer.start();
#endif
        // launch
        CU_CHECK(hipLaunchCooperativeKernel(
            (void *)tpr_ker, dim_grid, dim_block, kernel_args, shmem_size));

#ifdef TPR_PERF
        timer.stop_and_elapsed(elapsed);  // hipDeviceSynchronize called
        pmcpp::perf_time.push_back(elapsed);
    }
#endif

    CU_CHECK(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));

    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    CU_CHECK(hipFree(d_x));
    return;
}

/**
 * @brief launch configuration for tpr_ker
 * @details calculate suitable dimension and shared memory size for tpr_ker
 *
 * @param[in]  n     size of the equation
 * @param[in]  s     TPR parameter
 * @param[in]  dev   cuda device id
 * @return     [dim_grid, dim_block, shared_memory_size]
 */
std::tuple<dim3, dim3, size_t> PTPR_CU::tpr_launch_config(int n, int s,
                                                          int dev) {
    // check cooperative launch support
    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch,
                           dev);
    if (supportsCoopLaunch != 1) {
        printf("Cooperative launch not supported on dev %d.\n", dev);
        exit(EXIT_FAILURE);
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    // calculate dimension
    auto dim = n2dim(n, s, dev);
    auto dim_grid = dim[0];
    auto dim_block = dim[1];

    size_t shmem_size = 4 * dim_block.x * sizeof(float);
    assert(shmem_size <= deviceProp.sharedMemPerBlock);

    std::tuple<dim3, dim3, size_t> ret(dim_grid, dim_block, shmem_size);
    return ret;
}

/**
 * @brief Helper function for tpr_cu
 * @details calculate dimension for cuda kernel launch.
 *
 * @param[in]  n     size of the equation
 * @param[in]  s     TPR parameter
 * @param[in]  dev   cuda device id
 * @return     [dim_grid, dim_block]
 */
std::array<dim3, 2> PTPR_CU::n2dim(int n, int s, int dev) {
    assert(s > 0);
    assert(n >= s);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    auto max_tpb = deviceProp.maxThreadsPerBlock;

    if (s > max_tpb) {
        std::cerr << "TPR Parameter `s=" << s
                  << "` exceeds max threads per block: " << max_tpb << "\n";
        exit(EXIT_FAILURE);
    }

    dim3 dim_grid(n / s, 1, 1);  // we know `n >= s`
    dim3 dim_block(s, 1, 1);
    dim_grid.y = std::max(s / max_tpb, 1);

    return {dim_grid, dim_block};
}

void PTPR_CU::pcr_cu(float *a, float *c, float *rhs, float *x, int n) {
    int size = n * sizeof(float);

    // Device
    float *d_a, *d_c, *d_r;  // device copies of a, c, rhs
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));

    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    pcr_ker<<<1, n>>>(d_a, d_c, d_r, n);

    hipDeviceSynchronize();
    CU_CHECK(hipMemcpy(x, d_r, size, hipMemcpyDeviceToHost));

    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    return;
}
