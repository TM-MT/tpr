#include "reference_cusparse.cuh"

#ifdef TPR_PERF
#include "pm.cuh"
#endif

/**
 * @brief      check `hipError_t`
 *
 * @param      expr  The expression
 */
#define CU_CHECK(expr)                                                     \
    {                                                                      \
        hipError_t t = expr;                                              \
        if (t != hipSuccess) {                                            \
            fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", \
                    hipGetErrorString(t), t, __FILE__, __LINE__);         \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

/**
 * @brief      check `hipsparseStatus_t`
 *
 * @param      expr  The expression
 */
#define CUSP_CHECK(expr)                                                     \
    {                                                                        \
        hipsparseStatus_t t = expr;                                           \
        if (t != HIPSPARSE_STATUS_SUCCESS) {                                  \
            fprintf(stderr,                                                  \
                    "[CUSPARSE][Error] %s (error code: %d) at %s line %d\n", \
                    hipsparseGetErrorString(t), t, __FILE__, __LINE__);       \
            exit(EXIT_FAILURE);                                              \
        }                                                                    \
    }

/**
 * @brief      Helper function for cusparse::hipsparseSgtsv2_nopivot (CR+PCR)
 *
 * @param      a     { parameter_description }
 * @param      c     { parameter_description }
 * @param      rhs   The right hand side
 * @param      x     { parameter_description }
 * @param[in]  n     { parameter_description }
 */
void REFERENCE_CUSPARSE::solve(float *a, float *c, float *rhs, float *x,
                               int n) {
    /* step 3: prepare data in device */
    CU_CHECK(hipMemcpy(dl, a, size_of_mem, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d, diag, size_of_mem, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(du, c, size_of_mem, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(db, rhs, size_of_mem, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    // calculate the size of the buffer used in gtsv2_nopivot
    size_t pbuffsize;
    CUSP_CHECK(hipsparseSgtsv2_nopivot_bufferSizeExt(cusparseH, n, 1, dl, d, du,
                                                    db, n, &pbuffsize));

    CU_CHECK(hipMalloc((void **)&pBuffer, pbuffsize));

    hipDeviceSynchronize();

#ifdef TPR_PERF
    {
        time_ms elapsed = 0;
        pmcpp::DeviceTimer timer;
        timer.start();
#endif
        // execute
        CUSP_CHECK(
            hipsparseSgtsv2_nopivot(cusparseH, n, 1, dl, d, du, db, n, pBuffer));

#ifdef TPR_PERF
        timer.stop_and_elapsed(elapsed);  // hipDeviceSynchronize called
        pmcpp::perf_time.push_back(elapsed);
    }
#else
    hipDeviceSynchronize();
#endif

    CU_CHECK(hipMemcpy(x, db, size_of_mem, hipMemcpyDeviceToHost));
    CU_CHECK(hipFree(pBuffer));

    return;
}
