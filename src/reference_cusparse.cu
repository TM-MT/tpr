#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "reference_cusparse.cuh"

/**
 * @brief      check `hipError_t`
 *
 * @param      expr  The expression
 */
#define CU_CHECK(expr)                                                     \
    {                                                                      \
        hipError_t t = expr;                                              \
        if (t != hipSuccess) {                                            \
            fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", \
                    hipGetErrorString(t), t, __FILE__, __LINE__);         \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

/**
 * @brief      check `hipsparseStatus_t`
 *
 * @param      expr  The expression
 */
#define CUSP_CHECK(expr)                                                     \
    {                                                                        \
        hipsparseStatus_t t = expr;                                           \
        if (t != HIPSPARSE_STATUS_SUCCESS) {                                  \
            fprintf(stderr,                                                  \
                    "[CUSPARSE][Error] %s (error code: %d) at %s line %d\n", \
                    hipsparseGetErrorString(t), t, __FILE__, __LINE__);       \
            exit(EXIT_FAILURE);                                              \
        }                                                                    \
    }

/**
 * @brief      check `hipblasStatus_t`
 *
 * @param      expr  The expression
 */
#define CUBLAS_CHECK(expr)                                                 \
    {                                                                      \
        hipblasStatus_t t = expr;                                           \
        if (t != HIPBLAS_STATUS_SUCCESS) {                                  \
            fprintf(stderr,                                                \
                    "[CUBLAS][Error] (error code: %d) at %s line %d\n", t, \
                    __FILE__, __LINE__);                                   \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

/**
 * @brief      Helper function for cusparse::hipsparseSgtsv2_nopivot (CR+PCR)
 *
 * @param      a     { parameter_description }
 * @param      c     { parameter_description }
 * @param      rhs   The right hand side
 * @param      x     { parameter_description }
 * @param[in]  n     { parameter_description }
 */
void REFERENCE_CUSPARSE::ref_cusp(float *a, float *c, float *rhs, float *x,
                                  int n) {
    float *diag;
    diag = (float *)malloc(n * sizeof(float));
    for (int i = 0; i < n; i++) {
        diag[i] = 1.0;
    }

    hipsparseHandle_t cusparseH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    // device memory
    float *dl, *d, *du, *db;
    void *pBuffer;
    int size_of_mem = n * sizeof(float);

    /* step 1: create cusparse/cublas handle, bind a stream */
    CU_CHECK(hipStreamCreateWithFlags(&stream, hipStreamDefault));

    CUSP_CHECK(hipsparseCreate(&cusparseH));

    CUSP_CHECK(hipsparseSetStream(cusparseH, stream));
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: allocate device memory */
    CU_CHECK(hipMalloc((void **)&dl, size_of_mem));
    CU_CHECK(hipMalloc((void **)&d, size_of_mem));
    CU_CHECK(hipMalloc((void **)&du, size_of_mem));
    CU_CHECK(hipMalloc((void **)&db, size_of_mem));

    /* step 3: prepare data in device */
    CU_CHECK(hipMemcpy(dl, a, size_of_mem, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d, diag, size_of_mem, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(du, c, size_of_mem, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(db, rhs, size_of_mem, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    // calculate the size of the buffer used in gtsv2_nopivot
    size_t pbuffsize;
    CUSP_CHECK(hipsparseSgtsv2_nopivot_bufferSizeExt(cusparseH, n, 1, dl, d, du,
                                                    db, n, &pbuffsize));

    CU_CHECK(hipMalloc((void **)&pBuffer, pbuffsize));

    hipDeviceSynchronize();

    // execute
    CUSP_CHECK(
        hipsparseSgtsv2_nopivot(cusparseH, n, 1, dl, d, du, db, n, pBuffer));

    hipDeviceSynchronize();

    CU_CHECK(hipMemcpy(x, db, size_of_mem, hipMemcpyDeviceToHost));

    // free
    CU_CHECK(hipFree(pBuffer));
    CU_CHECK(hipFree(dl));
    CU_CHECK(hipFree(d));
    CU_CHECK(hipFree(du));
    CU_CHECK(hipFree(db));
    if (cusparseH) hipsparseDestroy(cusparseH);
    if (cublasH) hipblasDestroy(cublasH);
    if (stream) hipStreamDestroy(stream);

    hipDeviceReset();

    free(diag);
    return;
}
