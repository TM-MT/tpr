#include "reference_cusparse.cuh"

#ifdef TPR_PERF
#include "pm.cuh"
#endif

/**
 * @brief      check `hipError_t`
 *
 * @param      expr  The expression
 */
#define CU_CHECK(expr)                                                     \
    {                                                                      \
        hipError_t t = expr;                                              \
        if (t != hipSuccess) {                                            \
            fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", \
                    hipGetErrorString(t), t, __FILE__, __LINE__);         \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

/**
 * @brief      check `hipsparseStatus_t`
 *
 * @param      expr  The expression
 */
#define CUSP_CHECK(expr)                                                     \
    {                                                                        \
        hipsparseStatus_t t = expr;                                           \
        if (t != HIPSPARSE_STATUS_SUCCESS) {                                  \
            fprintf(stderr,                                                  \
                    "[CUSPARSE][Error] %s (error code: %d) at %s line %d\n", \
                    hipsparseGetErrorString(t), t, __FILE__, __LINE__);       \
            exit(EXIT_FAILURE);                                              \
        }                                                                    \
    }
/**
 * @brief      check `hipblasStatus_t`
 *
 * @param      expr  The expression
 */
#define CUBLAS_CHECK(expr)                                                 \
    {                                                                      \
        hipblasStatus_t t = expr;                                           \
        if (t != HIPBLAS_STATUS_SUCCESS) {                                  \
            fprintf(stderr,                                                \
                    "[CUBLAS][Error] (error code: %d) at %s line %d\n", t, \
                    __FILE__, __LINE__);                                   \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

REFERENCE_CUSPARSE::REFERENCE_CUSPARSE(int n) {
    diag = (float *)malloc(n * sizeof(float));
    for (int i = 0; i < n; i++) {
        diag[i] = 1.0;
    }

    size_of_mem = n * sizeof(float);

    /* step 1: create cusparse/cublas handle, bind a stream */
    CU_CHECK(hipStreamCreateWithFlags(&stream, hipStreamDefault));

    CUSP_CHECK(hipsparseCreate(&cusparseH));

    CUSP_CHECK(hipsparseSetStream(cusparseH, stream));
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: allocate device memory */
    CU_CHECK(hipMalloc((void **)&dl, size_of_mem));
    CU_CHECK(hipMalloc((void **)&d, size_of_mem));
    CU_CHECK(hipMalloc((void **)&du, size_of_mem));
    CU_CHECK(hipMalloc((void **)&db, size_of_mem));
}

REFERENCE_CUSPARSE::~REFERENCE_CUSPARSE() {
    // free
    CU_CHECK(hipFree(dl));
    CU_CHECK(hipFree(d));
    CU_CHECK(hipFree(du));
    CU_CHECK(hipFree(db));
    if (cusparseH) hipsparseDestroy(cusparseH);
    if (cublasH) hipblasDestroy(cublasH);
    if (stream) hipStreamDestroy(stream);

    hipDeviceReset();

    free(diag);
}

/**
 * @brief      Helper function for cusparse::hipsparseSgtsv2_nopivot (CR+PCR)
 *
 * @param      a     { parameter_description }
 * @param      c     { parameter_description }
 * @param      rhs   The right hand side
 * @param      x     { parameter_description }
 * @param[in]  n     { parameter_description }
 */
void REFERENCE_CUSPARSE::solve(float *a, float *c, float *rhs, float *x,
                               int n) {
    /* step 3: prepare data in device */
    CU_CHECK(hipMemcpy(dl, a, size_of_mem, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d, diag, size_of_mem, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(du, c, size_of_mem, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(db, rhs, size_of_mem, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    // calculate the size of the buffer used in gtsv2_nopivot
    size_t pbuffsize;
    CUSP_CHECK(hipsparseSgtsv2_nopivot_bufferSizeExt(cusparseH, n, 1, dl, d, du,
                                                    db, n, &pbuffsize));

    CU_CHECK(hipMalloc((void **)&pBuffer, pbuffsize));

    hipDeviceSynchronize();

#ifdef TPR_PERF
    {
        time_ms elapsed = 0;
        pmcpp::DeviceTimer timer;
        timer.start();
#endif
        // execute
        CUSP_CHECK(
            hipsparseSgtsv2_nopivot(cusparseH, n, 1, dl, d, du, db, n, pBuffer));

#ifdef TPR_PERF
        timer.stop_and_elapsed(elapsed);  // hipDeviceSynchronize called
        pmcpp::perf_time.push_back(elapsed);
    }
#else
    hipDeviceSynchronize();
#endif

    CU_CHECK(hipMemcpy(x, db, size_of_mem, hipMemcpyDeviceToHost));
    CU_CHECK(hipFree(pBuffer));

    return;
}
