#include "hip/hip_runtime.h"
#include "ptpr.cuh"
#include "main.hpp"
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

using namespace TPR_CU;



__global__ void tpr_ker(float *a, float *c, float *rhs, float *x, int n, int s) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int st = idx / s * s;
    int ed = st + s - 1;

    Equation eq;
    eq.a = a;
    eq.c = c;
    eq.rhs = rhs;
    eq.x = x;

    TPR_Params params;
    params.n = n;
    params.s = s;
    params.idx = idx;
    params.st = st;
    params.ed = ed;

    cg::thread_block tb = cg::this_thread_block();

    float tmp_aa, tmp_cc, tmp_rr;
    // bkups, .x -> a, .y -> c, .z -> rhs
    float3 bkup_st, bkup_ed;

    tpr_st1_ker(tb, eq, params);

    tpr_inter(tb, eq, &bkup_st, params);

    tb.sync();

    tpr_inter_global(tb, eq, &bkup_ed, params);

    tb.sync();

    // PCR
    for (int p = static_cast<int>(log2f(static_cast<double>(s))) + 1;
         p <= static_cast<int>(log2f(static_cast<double>(n)));
         p++)
    {
        if (idx < n && idx == ed) {
            // reduction
            int u = 1 << (p - 1); // offset
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < 0) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = a[lidx];
                ckl = c[lidx];
                rkl = rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx >= n) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = a[ridx];
                ckr = c[ridx];
                rkr = rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * a[idx] - akr * c[idx]);

            tmp_aa = - inv_diag_k * akl* a[idx];
            tmp_cc = - inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);
        }

        tb.sync();

        if (idx < n && idx == ed) {
        // copy back
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;
        }

        tb.sync();
    }

    tb.sync();


    tpr_st2_copyback(tb, rhs, x, n, s);
    tb.sync();
    // stage 3
    if (idx < n && idx == st) {
        a[idx] = bkup_st.x;
        c[idx] = bkup_st.y;
        rhs[idx] = bkup_st.z;
    }

    // should be same condition as tpr_inter_global
    if (idx < n && idx == ed) {
        a[idx] = bkup_ed.x;
        c[idx] = bkup_ed.y;
        rhs[idx] = bkup_ed.z;
    }

    tb.sync();

    tpr_st3_ker(tb, eq, params);
 
    return ;
}


// stage 1
__device__ void tpr_st1_ker(cg::thread_block &tb, Equation eq, TPR_Params const& params){
    int idx = params.idx;
    int n = params.n, s = params.s;
    int st = params.st, ed = params.ed;
    float tmp_aa, tmp_cc, tmp_rr;

    for (int p = 1; p <= static_cast<int>(log2f(static_cast<double>(s))); p++) {
        if (idx < n) {
            // reduction
            int u = 1 << (p - 1); // offset
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < st) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = eq.a[lidx];
                ckl = eq.c[lidx];
                rkl = eq.rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx > ed) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = eq.a[ridx];
                ckr = eq.c[ridx];
                rkr = eq.rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * eq.a[idx] - akr * eq.c[idx]);

            tmp_aa = - inv_diag_k * akl* eq.a[idx];
            tmp_cc = - inv_diag_k * ckr * eq.c[idx];
            tmp_rr = inv_diag_k * (eq.rhs[idx] - rkl * eq.a[idx] - rkr * eq.c[idx]);
        }

        tb.sync();

        if (idx < n) {
            // copy back
            eq.a[idx] = tmp_aa;
            eq.c[idx] = tmp_cc;
            eq.rhs[idx] = tmp_rr;
        }

        tb.sync();
    }
}

// TPR Intermidiate stage 1
// Update E_{st} by E_{ed}
__device__ void tpr_inter(cg::thread_block &tb, Equation eq, float3 *bkup, TPR_Params const& params){
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    float tmp_aa, tmp_cc, tmp_rr;

    if (idx < params.n && idx == params.st) {
        int k = params.st, kr = params.ed;
        float ak = eq.a[k], akr = eq.a[kr];
        float ck = eq.c[k], ckr = eq.c[kr];
        float rhsk = eq.rhs[k], rhskr = eq.rhs[kr];

        float inv_diag_k = 1.0 / (1.0 - akr * ck);

        tmp_aa = inv_diag_k * ak;
        tmp_cc = -inv_diag_k * ckr * ck;
        tmp_rr = inv_diag_k * (rhsk - rhskr * ck);

        // idx == st
        bkup->x = eq.a[idx];
        bkup->y = eq.c[idx];
        bkup->z = eq.rhs[idx];

        eq.a[idx] = tmp_aa;
        eq.c[idx] = tmp_cc;
        eq.rhs[idx] = tmp_rr;
    }
}

// Update E_{st-1} by E_{st}
__device__ void tpr_inter_global(cg::thread_block &tb, Equation eq, float3 *bkup, TPR_Params const& params) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int ed = params.ed;

    if (idx < params.n - 1 && idx == ed) {
        int k = idx, kr = idx+1; // (k, kr) = (st-1, st)
        float ak = eq.a[k], akr = eq.a[kr];
        float ck = eq.c[k], ckr = eq.c[kr];
        float rhsk = eq.rhs[k], rhskr = eq.rhs[kr];
        float inv_diag_k = 1.0 / (1.0 - akr * ck);

        bkup->x = eq.a[idx];
        bkup->y = eq.c[idx];
        bkup->z = eq.rhs[idx];

        eq.a[k] = inv_diag_k * ak;
        eq.c[k] = -inv_diag_k * ckr * ck;
        eq.rhs[k] = inv_diag_k * (rhsk - rhskr * ck);
    } else if (idx == params.n - 1) {
        bkup->x = eq.a[idx];
        bkup->y = eq.c[idx];
        bkup->z = eq.rhs[idx];
    }
}



// copy the answer from stage 2 PCR
__device__ void tpr_st2_copyback(cg::thread_block &tb, float *rhs, float *x, int n, int s) {
	int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int st = idx / s * s;
    int ed = st + s - 1;

    if (idx < n && idx == ed) {
        x[idx] = rhs[idx];
    }
}

__device__ void tpr_st3_ker(cg::thread_block &tb, Equation eq, TPR_Params const& params) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int st = params.st;
    int ed = params.ed;
    int n = params.n;

   if (idx < n) {
        int lidx = max(0, st - 1);

        float key = 1.0 / eq.c[ed] * (eq.rhs[ed] - eq.a[ed] * eq.x[lidx] - eq.x[ed]);
        if (eq.c[ed] == 0.0) {
            key = 0.0;
        }

        eq.x[idx] = eq.rhs[idx] - eq.a[idx] * eq.x[lidx] - eq.c[idx] * key;
    }
    return ;
}


__global__ void pcr_ker(float *a, float *c, float *rhs, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp_aa, tmp_cc, tmp_rr;

    if (idx < n) {
        for (int p = 1; p <= static_cast<int>(log2f(static_cast<double>(n))); p++) {
            // reduction
            int u = 1 << (p - 1); // offset
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < 0) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = a[lidx];
                ckl = c[lidx];
                rkl = rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx >= n) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = a[ridx];
                ckr = c[ridx];
                rkr = rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * a[idx] - akr * c[idx]);

            tmp_aa = - inv_diag_k * akl* a[idx];
            tmp_cc = - inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);

            __syncthreads();

            // copy back
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;

            __syncthreads();
        }
    }
}



#define CU_CHECK( expr ) { hipError_t t = expr;\
    if (t != hipSuccess) {\
        fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", hipGetErrorString(t), t, __FILE__, __LINE__); \
        exit(EXIT_FAILURE);\
    } \
}


int main() {
    int n = 1024;
    struct TRIDIAG_SYSTEM *sys = (struct TRIDIAG_SYSTEM *)malloc(sizeof(struct TRIDIAG_SYSTEM));
    setup(sys, n);
    for (int s = 128; s <= n; s *= 2) {
        assign(sys);
        tpr_cu(sys->a, sys->c, sys->rhs, n, s);
    }

    assign(sys);
    pcr_cu(sys->a, sys->c, sys->rhs, n);

    clean(sys);
    free(sys);

}


int setup(struct TRIDIAG_SYSTEM *sys, int n) {
    sys->a = (real *)malloc(n * sizeof(real));
    sys->diag = (real *)malloc(n * sizeof(real));
    sys->c = (real *)malloc(n * sizeof(real));
    sys->rhs = (real *)malloc(n * sizeof(real));
    sys->n = n;

    return sys_null_check(sys);
}

int assign(struct TRIDIAG_SYSTEM *sys) {
    int n = sys->n;
    for (int i = 0; i < n; i++) {
        sys->a[i] = -1.0/6.0;
        sys->c[i] = -1.0/6.0;
        sys->diag[i] = 1.0;
        sys->rhs[i] = 1.0 * (i+1);
    }
    sys->a[0] = 0.0;
    sys->c[n-1] = 0.0;

    return 0;
}



int clean(struct TRIDIAG_SYSTEM *sys) {
    for (auto p: { sys->a, sys->diag, sys->c, sys->rhs }) {
        free(p);
    }

    sys->a = nullptr;
    sys->diag = nullptr;
    sys->c = nullptr;
    sys->rhs = nullptr;

    return 0;
}


bool sys_null_check(struct TRIDIAG_SYSTEM *sys) {
    for (auto p: { sys->a, sys->diag, sys->c, sys->rhs }) {
        if (p == nullptr) {
            return false;
        }
    }
    return true;
}



void tpr_cu(float *a, float *c, float *rhs, int n, int s) {
    int size = n * sizeof(float);
    // Host
    float *x;

    x = (float*)malloc(size);

    // Device
    float *d_a, *d_c, *d_r;   // device copies of a, c, rhs
    float *d_x;
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));
    CU_CHECK(hipMalloc((void **)&d_x, size));

    std::cerr << "TPR: s=" << s << "\n";
    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice)); 
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    // launch
    tpr_ker<<<n / s, s>>>(d_a, d_c, d_r, d_x, n, s);

    hipDeviceSynchronize();

    CU_CHECK(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        std::cout << x[i] << ", ";
    }
    std::cout << "\n";


    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    CU_CHECK(hipFree(d_x));
    free(x);
    return ;
}



void pcr_cu(float *a, float *c, float *rhs, int n) {
    int size = n * sizeof(float);
    // Host
    float *x;

    x = (float*)malloc(size);

    // Device
    float *d_a, *d_c, *d_r;   // device copies of a, c, rhs
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));

    std::cerr << "PCR\n";
    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    pcr_ker<<<1, n>>>(d_a, d_c, d_r, n);

    hipDeviceSynchronize();
    CU_CHECK(hipMemcpy(x, d_r, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        std::cout << x[i] << ", ";
    }
    std::cout << "\n";

    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    free(x);
    return ;
}
