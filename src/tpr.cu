#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include <iostream>

#include "main.hpp"
#include "tpr.cuh"

#ifdef TPR_PERF
#include "pm.cuh"
#endif

#if (__CUDACC_VER_MAJOR__ <= 11) && (__CUDACC_VER_MINOR__ < 4)
#pragma message("Using Experimental Features")
#define EXPERIMENTAL_ASYNC_COPY
#endif

namespace cg = cooperative_groups;
#ifdef EXPERIMENTAL_ASYNC_COPY
using namespace nvcuda::experimental;
#endif

using namespace TPR_CU;

/**
 * for dynamic shared memory use
 */
extern __shared__ float array[];

/**
 * @brief      TPR main kernel of TPR
 *
 * @param[in]  a        a[0:n] The subdiagonal elements of A. Assert a[0] == 0.0
 * @param[in]  c        c[0:n] The superdiagonal elements of A. Assert c[n-1] ==
 *                      0.0
 * @param[in]  rhs      rhs[0:n] The right-hand-side of the equation.
 * @param[out] x        x[0:n] for the solution
 * @param      pbuffer  Additional memory for Stage 2 use. pbuffer[0:4 * n / s]
 * @param[in]  n        The order of A. `n` should be power of 2
 * @param[in]  s        The parameter of TPR. Each block handles `s` equations.
 */
__global__ void TPR_CU::tpr_ker(float *a, float *c, float *rhs, float *x,
                                float *pbuffer, int n, int s) {
    cg::grid_group tg = cg::this_grid();
    cg::thread_block tb = cg::this_thread_block();
    assert(tg.is_valid());
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int st = idx / s * s;

    // local copy
    // sha[0:s], shc[0:s], shrhs[0:s]
    __shared__ float *sha, *shc, *shrhs;
    sha = (float *)array;
    shc = (float *)&array[s];
    shrhs = (float *)&array[2 * s];

    // make local copy on shared memory
#ifdef EXPERIMENTAL_ASYNC_COPY
    pipeline pipe;
    memcpy_async(sha[idx - st], a[idx], pipe);
    memcpy_async(shc[idx - st], c[idx], pipe);
    memcpy_async(shrhs[idx - st], rhs[idx], pipe);
#else
    cg::memcpy_async(tb, sha, &a[st], sizeof(float) * s);
    cg::memcpy_async(tb, shc, &c[st], sizeof(float) * s);
    cg::memcpy_async(tb, shrhs, &rhs[st], sizeof(float) * s);
#endif

    Equation eq;
    eq.a = sha;
    eq.c = shc;
    eq.rhs = shrhs;
    eq.x = x;

    TPR_Params params;
    params.n = n;
    params.s = s;
    params.m = n / s;
    params.idx = idx;
    params.st = st;
    params.ed = st + s - 1;

    // bkups for stage 3 use, .x -> a, .y -> c, .z -> rhs
    float3 bkup;

#ifdef EXPERIMENTAL_ASYNC_COPY
    pipe.commit_and_wait();
#else
    cg::wait(tb);
#endif

    // TPR Stage 1
    if (idx < n && idx % 2 == 0) {
        bkup.x = sha[idx - st];
        bkup.y = shc[idx - st];
        bkup.z = shrhs[idx - st];
    }

    tpr_st1_ker(tb, eq, params);

    if (idx < n && idx % 2 == 1) {
        bkup.x = sha[idx - st];
        bkup.y = shc[idx - st];
        bkup.z = shrhs[idx - st];
    }

    tb.sync();

    tpr_inter(tb, eq, params);

    // copy back
    // since stage 2 are global operations, eq.* should hold address in global
    // memory
    a[idx] = sha[idx - st];
    c[idx] = shc[idx - st];
    rhs[idx] = shrhs[idx - st];
    eq.a = a;
    eq.c = c;
    eq.rhs = rhs;

    // make sure stage 1 operations have done.
    tg.sync();

    tpr_st2_ker(tg, tb, eq, params, pbuffer);

    tg.sync();

#ifdef EXPERIMENTAL_ASYNC_COPY
    if (blockIdx.x == 0 && idx < params.m) {
        pipe.commit_and_wait();
    }
#else
    if (blockIdx.x == 0) {
        cg::wait(tb);
    }
#endif

    // TPR stage 3
    if (idx < n) {
        sha[idx - st] = bkup.x;
        shc[idx - st] = bkup.y;
        shrhs[idx - st] = bkup.z;
    }

    tb.sync();

    // tpr_st3_ker use shared memory
    eq.a = sha;
    eq.c = shc;
    eq.rhs = shrhs;
    tpr_st3_ker(tb, eq, params);

    return;
}

/**
 * @brief         TPR Stage 1
 *
 * @param         tb      cg::thread_block
 * @param[in,out] eq      Equation. `eq.a, eq.c, eq.rhs` should be address in
 *                        shared memory
 * @param[in]     params  The parameters of TPR
 */
__device__ void TPR_CU::tpr_st1_ker(cg::thread_block &tb, Equation &eq,
                                    TPR_Params const &params) {
    int idx = params.idx;
    int i = idx - params.st;
    int s = params.s;
    float tmp_aa, tmp_cc, tmp_rr;
    assert(__isShared((void *)eq.a));
    assert(__isShared((void *)eq.c));
    assert(__isShared((void *)eq.rhs));

    for (int p = 1; p <= static_cast<int>(log2f(static_cast<float>(s))); p++) {
        int p2k = 1 << p;
        bool select_idx =
            idx < params.n && ((i % p2k == 0) || ((i + 1) % p2k == 0));

        if (select_idx) {
            int u = 1 << (p - 1);  // offset

            // reduction
            float akl, ckl, rkl;
            {
                int lidx = i - u;
                if (lidx < 0) {
                    akl = -1.0f;
                    ckl = 0.0f;
                    rkl = 0.0f;
                } else {
                    akl = eq.a[lidx];
                    ckl = eq.c[lidx];
                    rkl = eq.rhs[lidx];
                }
            }
            float akr, ckr, rkr;
            {
                int ridx = i + u;
                if (ridx >= s) {
                    akr = 0.0f;
                    ckr = -1.0f;
                    rkr = 0.0f;
                } else {
                    akr = eq.a[ridx];
                    ckr = eq.c[ridx];
                    rkr = eq.rhs[ridx];
                }
            }

            float inv_diag_k = 1.0f / (1.0f - ckl * eq.a[i] - akr * eq.c[i]);

            tmp_aa = -inv_diag_k * akl * eq.a[i];
            tmp_cc = -inv_diag_k * ckr * eq.c[i];
            tmp_rr = inv_diag_k * (eq.rhs[i] - rkl * eq.a[i] - rkr * eq.c[i]);
        }

        tb.sync();

        if (select_idx) {
            // copy back
            eq.a[i] = tmp_aa;
            eq.c[i] = tmp_cc;
            eq.rhs[i] = tmp_rr;
        }

        tb.sync();
    }
}

/**
 * @brief         TPR Intermediate stage 1
 *
 *                Update E_{st} by E_{ed}
 *
 * @param         tb      cg::thread_block
 * @param[in,out] eq      Equation. `eq.a, eq.c, eq.rhs` should be address in
 *                        shared memory
 * @param[in]     params  The parameters of TPR
 */
__device__ void TPR_CU::tpr_inter(cg::thread_block &tb, Equation &eq,
                                  TPR_Params const &params) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    assert(__isShared((void *)eq.a));
    assert(__isShared((void *)eq.c));
    assert(__isShared((void *)eq.rhs));

    if ((idx < params.n) && (idx == params.st)) {
        int k = idx - params.st, kr = params.s - 1;
        float ak = eq.a[k], akr = eq.a[kr];
        float ck = eq.c[k], ckr = eq.c[kr];
        float rhsk = eq.rhs[k], rhskr = eq.rhs[kr];

        float inv_diag_k = 1.0f / (1.0f - akr * ck);

        eq.a[k] = inv_diag_k * ak;
        eq.c[k] = -inv_diag_k * ckr * ck;
        eq.rhs[k] = inv_diag_k * (rhsk - rhskr * ck);
    }
}

/**
 * @brief         TPR Intermediate stage GLOBAL
 *
 *                Update E_{st-1} by E_{st}
 *
 * @param         tb       cg::thread_block
 * @param[in,out] eq       Equation. `eq.a, eq.c, eq.rhs` should be address in
 *                         GLOBAL memory
 * @param[in]     params   The parameters of TPR
 * @param         pbuffer  Additional memory for stage 2 use.
 */
__device__ void TPR_CU::tpr_inter_global(cg::thread_block &tb, Equation &eq,
                                         TPR_Params const &params,
                                         float *pbuffer) {
    assert(__isGlobal((void *)eq.a));
    assert(__isGlobal((void *)eq.c));
    assert(__isGlobal((void *)eq.rhs));
    int idx = params.idx;
    int ed = params.ed;

    if ((idx < params.n - 1) && (idx == ed)) {  // Update E_{st-1} by E_{st}
        int dst = idx / params.s;
        int k = idx, kr = idx + 1;  // (k, kr) = (st-1, st)
        float ak = eq.a[k], akr = eq.a[kr];
        float ck = eq.c[k], ckr = eq.c[kr];
        float rhsk = eq.rhs[k], rhskr = eq.rhs[kr];
        float inv_diag_k = 1.0f / (1.0f - akr * ck);

        pbuffer[dst] = inv_diag_k * ak;                    // a[k]
        pbuffer[params.m + dst] = -inv_diag_k * ckr * ck;  // c[k]
        pbuffer[2 * params.m + dst] =
            inv_diag_k * (rhsk - rhskr * ck);  // rhs[k]
    } else if (idx == params.n - 1) {          // copy E_{n-1} to pbuffer
        pbuffer[params.m - 1] = eq.a[idx];
        pbuffer[2 * params.m - 1] = eq.c[idx];
        pbuffer[3 * params.m - 1] = eq.rhs[idx];
    }
}

/**
 * @brief      TPR Stage 2
 *
 *             Call CR
 *
 * @param      tg       cg::grid_group
 * @param      tb       cg::thread_group
 * @param[in]  eq       Equation. `eq.a, eq.c, eq.rhs` should be address in
 *                      GLOBAL memory
 * @param[in]  params   The parameters of TPR
 * @param      pbuffer  Additional global memory for stage 2
 */
__device__ void TPR_CU::tpr_st2_ker(cg::grid_group &tg, cg::thread_block &tb,
                                    Equation &eq, TPR_Params &params,
                                    float *pbuffer) {
    tpr_inter_global(tb, eq, params, pbuffer);

    tg.sync();

    if (blockIdx.x == 0) {
        int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
        int s = params.s, m = params.m;
        assert(m <= s);

        __shared__ float *sha, *shc, *shrhs, *shx;
        sha = (float *)array;
        shc = (float *)&array[s];
        shrhs = (float *)&array[2 * s];
        shx = (float *)&array[3 * s];

#ifdef EXPERIMENTAL_ASYNC_COPY
        pipeline pipe;
        if (idx < m) {
            memcpy_async(sha[idx - params.st], pbuffer[idx], pipe);
            memcpy_async(shc[idx - params.st], pbuffer[m + idx], pipe);
            memcpy_async(shrhs[idx - params.st], pbuffer[2 * m + idx], pipe);
            memcpy_async(shx[idx - params.st], pbuffer[3 * m + idx], pipe);
            pipe.commit_and_wait();
        }
#else
        cg::memcpy_async(tb, sha, &pbuffer[0], sizeof(float) * m);
        cg::memcpy_async(tb, shc, &pbuffer[m], sizeof(float) * m);
        cg::memcpy_async(tb, shrhs, &pbuffer[2 * m], sizeof(float) * m);
        cg::memcpy_async(tb, shx, &pbuffer[3 * m], sizeof(float) * m);
        cg::wait(tb);  // following `cr_thread_block()` needs sh*
#endif

        cr_thread_block(tb, sha, shc, shrhs, shx, m);

#ifdef EXPERIMENTAL_ASYNC_COPY
        if (idx < m) {
            memcpy_async(sha[idx - params.st], eq.a[idx], pipe);
            memcpy_async(shc[idx - params.st], eq.c[idx], pipe);
            memcpy_async(shrhs[idx - params.st], eq.rhs[idx], pipe);
        }
#else
        // we only modified first `m` elements.
        cg::memcpy_async(tb, sha, &eq.a[params.st], sizeof(float) * m);
        cg::memcpy_async(tb, shc, &eq.c[params.st], sizeof(float) * m);
        cg::memcpy_async(tb, shrhs, &eq.rhs[params.st], sizeof(float) * m);
#endif
        // copy back
        if (idx < m) {
            int dst = (idx + 1) * s - 1;
            assert(dst < params.n);
            eq.x[dst] = shx[idx];
        }
    }
}

/**
 * @brief         TPR Stage 3
 *
 * @param         tb      cg::thread_block
 * @param[in,out] eq      Equation. `eq.a, eq.c, eq.rhs` should be address in
 *                        shared memory, `eq.x` should be address in GLOBAL
 *                        memory
 * @param[in]     params  The parameters of TPR
 */
__device__ void TPR_CU::tpr_st3_ker(cg::thread_block &tb, Equation &eq,
                                    TPR_Params const &params) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int i = tb.thread_index().x;
    assert(__isShared((void *)eq.a));
    assert(__isShared((void *)eq.c));
    assert(__isShared((void *)eq.rhs));
    assert(__isGlobal((void *)eq.x));

    for (int p = static_cast<int>(log2f(static_cast<float>(params.s))) - 1;
         p >= 0; p--) {
        int u = 1 << p;

        if (idx < params.n && ((idx - params.st - u + 1) % (2 * u) == 0) &&
            ((idx - params.st - u + 1) >= 0)) {
            int lidx = idx - u;
            float x_u;
            if (lidx < 0) {
                x_u = 0.0f;
            } else {
                x_u = eq.x[lidx];
            }

            eq.x[idx] = eq.rhs[i] - eq.a[i] * x_u - eq.c[i] * eq.x[idx + u];
        }
        tb.sync();
    }
    return;
}

/**
 * @brief      CR
 *
 *             Solve A*x = B by CR, where A is an n-by-n tridiagonal matrix, B
 *             is the right-hand-side vector of legth n
 *
 * @note       Only works in a block.
 *
 * @param      tb    cg::thread_block
 * @param[in]  a     a[0:n] The subdiagonal elements of A. Assert a[0] == 0.0
 * @param[in]  c     c[0:n] The superdiagonal elements of A. Assert c[n-1] ==
 *                   0.0
 * @param[in]  rhs   rhs[0:n] The right-hand-side of the equation.
 * @param[out] x     x[0:n] for the solution
 * @param[in]  n     The order of A. `n` should be power of 2
 */
__global__ void TPR_CU::cr_ker(float *a, float *c, float *rhs, float *x,
                               int n) {
    cg::thread_block tb = cg::this_thread_block();
    cr_thread_block(tb, a, c, rhs, x, n);
}

/**
 * @brief      CR
 *
 *             Solve A*x = B by CR, where A is an n-by-n tridiagonal matrix, B
 *             is the right-hand-side vector of legth n
 *
 * @note       Only works in a block.
 *
 * @param      tb    cg::thread_block
 * @param[in]  a     a[0:n] The subdiagonal elements of A. Assert a[0] == 0.0
 * @param[in]  c     c[0:n] The superdiagonal elements of A. Assert c[n-1] ==
 *                   0.0
 * @param[in]  rhs   rhs[0:n] The right-hand-side of the equation.
 * @param[out] x     x[0:n] for the solution
 * @param[in]  n     The order of A. `n` should be power of 2
 */
__device__ void TPR_CU::cr_thread_block(cg::thread_block &tb, float *a,
                                        float *c, float *rhs, float *x, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int p = 0; p < static_cast<int>(log2f(static_cast<float>(n))) - 1;
         p++) {
        float tmp_aa, tmp_cc, tmp_rr;
        int u = 1 << p;  // offset
        int ux = 1 << (p + 1);
        bool condition =
            (idx < n) && ((idx - ux + 1) % ux == 0) && ((idx - ux + 1) >= 0);

        // reduction
        if (condition) {
            float akl, ckl, rkl;
            {
                int lidx = idx - u;
                if (lidx < 0) {
                    akl = -1.0f;
                    ckl = 0.0f;
                    rkl = 0.0f;
                } else {
                    akl = a[lidx];
                    ckl = c[lidx];
                    rkl = rhs[lidx];
                }
            }
            float akr, ckr, rkr;
            {
                int ridx = idx + u;
                if (ridx >= n) {
                    akr = 0.0f;
                    ckr = -1.0f;
                    rkr = 0.0f;
                } else {
                    akr = a[ridx];
                    ckr = c[ridx];
                    rkr = rhs[ridx];
                }
            }

            float inv_diag_k = 1.0f / (1.0f - ckl * a[idx] - akr * c[idx]);

            tmp_aa = -inv_diag_k * akl * a[idx];
            tmp_cc = -inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);
        }

        __syncthreads();

        if (condition) {
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;
        }

        __syncthreads();
    }

    if ((n > 1) && (idx == n / 2 - 1)) {
        int u = n / 2;
        float inv_det = 1.0f / (1.0f - c[idx] * a[idx + u]);

        x[idx] = (rhs[idx] - c[idx] * rhs[idx + u]) * inv_det;
        x[idx + u] = (rhs[idx + u] - rhs[idx] * a[idx + u]) * inv_det;
    } else if (n == 1) {
        x[0] = rhs[0];
    }

    __syncthreads();

    for (int p = static_cast<int>(log2f(static_cast<float>(n))) - 2; p >= 0;
         p--) {
        int u = 1 << p;
        int ux = 1 << (p + 1);

        if ((idx < n) && ((idx - u + 1) % ux == 0) && (idx - u + 1 >= 0)) {
            int lidx = idx - u;
            float x_u;
            if (lidx < 0) {
                x_u = 0.0f;
            } else {
                x_u = x[lidx];
            }
            x[idx] = rhs[idx] - a[idx] * x_u - c[idx] * x[idx + u];
        }

        __syncthreads();
    }
    return;
}

#define CU_CHECK(expr)                                                     \
    {                                                                      \
        hipError_t t = expr;                                              \
        if (t != hipSuccess) {                                            \
            fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", \
                    hipGetErrorString(t), t, __FILE__, __LINE__);         \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

/**
 * @brief      Helper function for tpr_cu
 *
 *             Solve A*x = B by TPR, where A is an n-by-n tridiagonal matrix, B
 *             is the right-hand-side vector of legth n
 *
 * @note       assert the diagonal elements of A are 1.0
 *
 * 1. check if device support cooperative launch
 * 2. allocate device memory for compute
 * 3. launch kernel `TPR_CU::tpr_ker`
 * 4. copy the answer from device to host
 * 5. free device memory
 *
 * @param[in]  a     a[0:n] The subdiagonal elements of A. Assert a[0] == 0.0
 * @param[in]  c     c[0:n] The superdiagonal elements of A. Assert c[n-1] ==
 *                   0.0
 * @param[in]  rhs   rhs[0:n] The right-hand-side of the equation.
 * @param[out] x     x[0:n] for the solution
 * @param[in]  n     The order of A. `n` should be power of 2
 * @param[in]  s     The parameter of TPR. Each block handles `s` equations.
 */
void TPR_CU::tpr_cu(float *a, float *c, float *rhs, float *x, int n, int s) {
    if (n / s > s) {
        fprintf(stderr, "Not supported parameters given. (n, s)=(%d, %d)\n", n,
                s);
        return;
    }
    int dev = 0;
    int size = n * sizeof(float);

    // Device
    float *d_a, *d_c, *d_r;  // device copies of a, c, rhs
    float *d_x, *d_pbuffer;
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));
    CU_CHECK(hipMalloc((void **)&d_x, size));
    CU_CHECK(hipMalloc((void **)&d_pbuffer, 4 * n / s * sizeof(float)));

    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    // launch configuration
    void *kernel_args[] = {&d_a, &d_c, &d_r, &d_x, &d_pbuffer, &n, &s};
    auto config = tpr_launch_config(n, s, dev);
    // auto [dim_grid, dim_block, shmem_size] = rhs; not supported
    auto dim_grid = std::get<0>(config);
    auto dim_block = std::get<1>(config);
    auto shmem_size = std::get<2>(config);

#ifdef TPR_PERF
    {
        time_ms elapsed = 0;
        pmcpp::DeviceTimer timer;
        timer.start();
#endif
        // launch
        CU_CHECK(hipLaunchCooperativeKernel(
            (void *)tpr_ker, dim_grid, dim_block, kernel_args, shmem_size));
#ifdef TPR_PERF
        timer.stop_and_elapsed(elapsed);  // hipDeviceSynchronize called
        pmcpp::perf_time.push_back(elapsed);
    }
#else
    hipDeviceSynchronize();
#endif

    CU_CHECK(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));

    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    CU_CHECK(hipFree(d_x));
    CU_CHECK(hipFree(d_pbuffer));
    return;
}

/**
 * @brief launch configuration for tpr_ker
 * @details calculate suitable dimension and shared memory size for tpr_ker
 *
 * @param[in]  n     size of the equation
 * @param[in]  s     TPR parameter
 * @param[in]  dev   cuda device id
 * @return     [dim_grid, dim_block, shared_memory_size]
 */
std::tuple<dim3, dim3, size_t> TPR_CU::tpr_launch_config(int n, int s,
                                                         int dev) {
    // check cooperative launch support
    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch,
                           dev);
    if (supportsCoopLaunch != 1) {
        printf("Cooperative launch not supported on dev %d.\n", dev);
        exit(EXIT_FAILURE);
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    // calculate dimension
    auto dim = n2dim(n, s, dev);
    auto dim_grid = dim[0];
    auto dim_block = dim[1];

    size_t shmem_size = 4 * dim_block.x * sizeof(float);
    assert(shmem_size <= deviceProp.sharedMemPerBlock);

    std::tuple<dim3, dim3, size_t> ret(dim_grid, dim_block, shmem_size);
    return ret;
}

/**
 * @brief Helper function for tpr_cu
 * @details calculate dimension for cuda kernel launch.
 *
 * @param[in]  n     size of the equation
 * @param[in]  s     TPR parameter
 * @param[in]  dev   cuda device id
 * @return     [dim_grid, dim_block]
 */
std::array<dim3, 2> TPR_CU::n2dim(int n, int s, int dev) {
    assert(s > 0);
    assert(n >= s);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    auto max_tpb = deviceProp.maxThreadsPerBlock;

    if (s > max_tpb) {
        std::cerr << "TPR Parameter `s=" << s
                  << "` exceeds max threads per block: " << max_tpb << "\n";
        exit(EXIT_FAILURE);
    }

    dim3 dim_grid(n / s, 1, 1);  // we know `n >= s`
    dim3 dim_block(s, 1, 1);
    dim_grid.y = std::max(s / max_tpb, 1);

    return {dim_grid, dim_block};
}

void TPR_CU::cr_cu(float *a, float *c, float *rhs, float *x, int n) {
    int size = n * sizeof(float);

    // Device
    float *d_a, *d_c, *d_r, *d_x;  // device copies of a, c, rhs
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));
    CU_CHECK(hipMalloc((void **)&d_x, size));

    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    cr_ker<<<1, n>>>(d_a, d_c, d_r, d_x, n);

    hipDeviceSynchronize();
    CU_CHECK(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));

    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    CU_CHECK(hipFree(d_x));
    return;
}

#ifdef _REAL_IS_DOUBLE_
void TPR_CU::tpr_cu(double *a, double *c, double *rhs, double *x, int n,
                    int s) {
#pragma message " NOT IMPLEMENT."
}

void TPR_CU::cr_cu(double *a, double *c, double *rhs, double *x, int n) {
#pragma message " NOT IMPLEMENT."
}
#endif
