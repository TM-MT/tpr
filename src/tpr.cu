#include "hip/hip_runtime.h"
#include "tpr.cuh"
#include "main.hpp"
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

using namespace TPR_CU;



__global__ void tpr_ker(float *a, float *c, float *rhs, float *x, int n, int s) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int st = idx / s * s;
    int ed = st + s - 1;

    Equation eq;
    eq.a = a;
    eq.c = c;
    eq.rhs = rhs;
    eq.x = x;

    TPR_Params params;
    params.n = n;
    params.s = s;
    params.idx = idx;
    params.st = st;
    params.ed = ed;

    cg::thread_block tb = cg::this_thread_block();

    float tmp_aa, tmp_cc, tmp_rr;
    // bkups, .x -> a, .y -> c, .z -> rhs
    float3 bkup;

    if (idx < n && idx % 2 == 0) {
        bkup.x = a[idx];
        bkup.y = c[idx];
        bkup.z = rhs[idx];
    }

    tpr_st1_ker(tb, eq, params);


    if (idx < n && idx % 2 == 1) {
        bkup.x = a[idx];
        bkup.y = c[idx];
        bkup.z = rhs[idx];
    }

    tpr_inter(tb, eq, NULL, params);

    tb.sync();

    tpr_inter_global(tb, eq, NULL, params);

    tb.sync();

    // PCR
    for (int p = static_cast<int>(log2f(static_cast<double>(s))) + 1;
         p <= static_cast<int>(log2f(static_cast<double>(n)));
         p++)
    {
        if (idx < n && idx == ed) {
            // reduction
            int u = 1 << (p - 1); // offset
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < 0) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = a[lidx];
                ckl = c[lidx];
                rkl = rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx >= n) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = a[ridx];
                ckr = c[ridx];
                rkr = rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * a[idx] - akr * c[idx]);

            tmp_aa = - inv_diag_k * akl* a[idx];
            tmp_cc = - inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);
        }

        tb.sync();

        if (idx < n && idx == ed) {
        // copy back
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;
        }

        tb.sync();
    }

    tb.sync();


    tpr_st2_copyback(tb, rhs, x, n, s);
    tb.sync();
    // stage 3
    if (idx < n) {
        a[idx] = bkup.x;
        c[idx] = bkup.y;
        rhs[idx] = bkup.z;
    }

    tb.sync();

    tpr_st3_ker(tb, eq, params);
 
    return ;
}


// stage 1
__device__ void tpr_st1_ker(cg::thread_block &tb, Equation eq, TPR_Params const& params){
    int idx = params.idx;
    int n = params.n, s = params.s;
    int st = params.st, ed = params.ed;
    float tmp_aa, tmp_cc, tmp_rr;

    for (int p = 1; p <= static_cast<int>(log2f(static_cast<double>(s))); p++) {
        int u = 1 << (p - 1); // offset
        int p2k = 1 << p;
        if (idx < n && (((idx - st) % p2k == 0) || ((idx - st + 1) % p2k == 0))) {
            // reduction
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < st) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = eq.a[lidx];
                ckl = eq.c[lidx];
                rkl = eq.rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx > ed) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = eq.a[ridx];
                ckr = eq.c[ridx];
                rkr = eq.rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * eq.a[idx] - akr * eq.c[idx]);

            tmp_aa = - inv_diag_k * akl* eq.a[idx];
            tmp_cc = - inv_diag_k * ckr * eq.c[idx];
            tmp_rr = inv_diag_k * (eq.rhs[idx] - rkl * eq.a[idx] - rkr * eq.c[idx]);
        }

        tb.sync();

        if (idx < n) {
            // copy back
            eq.a[idx] = tmp_aa;
            eq.c[idx] = tmp_cc;
            eq.rhs[idx] = tmp_rr;
        }

        tb.sync();
    }
}

// TPR Intermidiate stage 1
// Update E_{st} by E_{ed}
__device__ void tpr_inter(cg::thread_block &tb, Equation eq, float3 *bkup, TPR_Params const& params){
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    float tmp_aa, tmp_cc, tmp_rr;

    if ((idx < params.n) && (idx == params.st)) {
        int k = params.st, kr = params.ed;
        float ak = eq.a[k], akr = eq.a[kr];
        float ck = eq.c[k], ckr = eq.c[kr];
        float rhsk = eq.rhs[k], rhskr = eq.rhs[kr];

        float inv_diag_k = 1.0 / (1.0 - akr * ck);

        tmp_aa = inv_diag_k * ak;
        tmp_cc = -inv_diag_k * ckr * ck;
        tmp_rr = inv_diag_k * (rhsk - rhskr * ck);

        eq.a[idx] = tmp_aa;
        eq.c[idx] = tmp_cc;
        eq.rhs[idx] = tmp_rr;
    }
}

// Update E_{st-1} by E_{st}
__device__ void tpr_inter_global(cg::thread_block &tb, Equation eq, float3 *bkup, TPR_Params const& params) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int ed = params.ed;

    if ((idx < params.n - 1) && (idx == ed)) {
        int k = idx, kr = idx+1; // (k, kr) = (st-1, st)
        float ak = eq.a[k], akr = eq.a[kr];
        float ck = eq.c[k], ckr = eq.c[kr];
        float rhsk = eq.rhs[k], rhskr = eq.rhs[kr];
        float inv_diag_k = 1.0 / (1.0 - akr * ck);

        eq.a[k] = inv_diag_k * ak;
        eq.c[k] = -inv_diag_k * ckr * ck;
        eq.rhs[k] = inv_diag_k * (rhsk - rhskr * ck);
    }
}



// copy the answer from stage 2 PCR
__device__ void tpr_st2_copyback(cg::thread_block &tb, float *rhs, float *x, int n, int s) {
	int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int st = idx / s * s;
    int ed = st + s - 1;

    if (idx < n && idx == ed) {
        x[idx] = rhs[idx];
    }
}

__device__ void tpr_st3_ker(cg::thread_block &tb, Equation eq, TPR_Params const& params) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int st = params.st;
    int n = params.n, s = params.s;

    for (int p = static_cast<int>(log2f(static_cast<double>(s))) - 1; p >= 0; p--) {
        int u = 1 << p;

        if (idx < n 
            && ((idx - st - u + 1) % (2 * u) == 0) 
            && ((idx - st - u + 1) >= 0)) {
            int lidx = idx - u;
            float x_u;
            if (lidx < 0) {
                x_u = 0.0;
            } else {
                x_u = eq.x[lidx];
            }

            eq.x[idx] = eq.rhs[idx] - eq.a[idx] * x_u - eq.c[idx] * eq.x[idx+u];
        }
        tb.sync();
    }
    return ;
}


__global__ void cr_ker(float *a, float *c, float *rhs, float *x, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp_aa, tmp_cc, tmp_rr;

    for (int p = 0; p < static_cast<int>(log2f(static_cast<double>(n))) - 1; p++) {
        int u = 1 << p; // offset
        int ux = 1 << (p + 1);
        bool condition = (idx < n) 
            && ((idx - ux + 1) % ux == 0)
            && ((idx - ux + 1) >= 0);

        // reduction
        if (condition) {
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < 0) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = a[lidx];
                ckl = c[lidx];
                rkl = rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx >= n) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = a[ridx];
                ckr = c[ridx];
                rkr = rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * a[idx] - akr * c[idx]);

            tmp_aa = - inv_diag_k * akl* a[idx];
            tmp_cc = - inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);
        }

        __syncthreads();

        if (condition) {
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;
        }

        __syncthreads();
    }

    if ((n > 1) && (idx == n / 2 - 1)) {
        int u = n / 2;
        float inv_det = 1.0 / (1.0 - c[idx]*a[idx+u]);

        x[idx] = (rhs[idx] - c[idx]*rhs[idx+u]) * inv_det;
        x[idx+u] =  (rhs[idx+u] - rhs[idx]*a[idx+u]) * inv_det;
    }

    __syncthreads();

    for (int p = static_cast<int>(log2f(static_cast<double>(n)))-2; p >= 0; p--) {
        int u = 1 << p;
        int ux = 1 << (p+1);
        
        if ((idx < n)
            && ((idx - u + 1) % ux == 0)
            && (idx - u + 1 >= 0))
        {
            int lidx = idx - u;
            float x_u;
            if (lidx < 0) {
                x_u = 0.0;
            } else {
                x_u = x[lidx];
            }
            x[idx] = rhs[idx] - a[idx] * x_u - c[idx] * x[idx+u];
        }

        __syncthreads();
    }
    return ;
}



#define CU_CHECK( expr ) { hipError_t t = expr;\
    if (t != hipSuccess) {\
        fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", hipGetErrorString(t), t, __FILE__, __LINE__); \
        exit(EXIT_FAILURE);\
    } \
}


int main() {
    int n = 1024;
    struct TRIDIAG_SYSTEM *sys = (struct TRIDIAG_SYSTEM *)malloc(sizeof(struct TRIDIAG_SYSTEM));
    setup(sys, n);
    for (int s = 256; s <= n; s *= 2) {
        assign(sys);
        tpr_cu(sys->a, sys->c, sys->rhs, n, s);
    }

    assign(sys);
    cr_cu(sys->a, sys->c, sys->rhs, n);

    clean(sys);
    free(sys);

}


int setup(struct TRIDIAG_SYSTEM *sys, int n) {
    sys->a = (real *)malloc(n * sizeof(real));
    sys->diag = (real *)malloc(n * sizeof(real));
    sys->c = (real *)malloc(n * sizeof(real));
    sys->rhs = (real *)malloc(n * sizeof(real));
    sys->n = n;

    return sys_null_check(sys);
}

int assign(struct TRIDIAG_SYSTEM *sys) {
    int n = sys->n;
    for (int i = 0; i < n; i++) {
        sys->a[i] = -1.0/6.0;
        sys->c[i] = -1.0/6.0;
        sys->diag[i] = 1.0;
        sys->rhs[i] = 1.0 * (i+1);
    }
    sys->a[0] = 0.0;
    sys->c[n-1] = 0.0;

    return 0;
}



int clean(struct TRIDIAG_SYSTEM *sys) {
    for (auto p: { sys->a, sys->diag, sys->c, sys->rhs }) {
        free(p);
    }

    sys->a = nullptr;
    sys->diag = nullptr;
    sys->c = nullptr;
    sys->rhs = nullptr;

    return 0;
}


bool sys_null_check(struct TRIDIAG_SYSTEM *sys) {
    for (auto p: { sys->a, sys->diag, sys->c, sys->rhs }) {
        if (p == nullptr) {
            return false;
        }
    }
    return true;
}



void tpr_cu(float *a, float *c, float *rhs, int n, int s) {
    int size = n * sizeof(float);
    // Host
    float *x;

    x = (float*)malloc(size);

    // Device
    float *d_a, *d_c, *d_r;   // device copies of a, c, rhs
    float *d_x;
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));
    CU_CHECK(hipMalloc((void **)&d_x, size));

    std::cerr << "TPR: s=" << s << "\n";
    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice)); 
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    // launch
    tpr_ker<<<n / s, s>>>(d_a, d_c, d_r, d_x, n, s);

    hipDeviceSynchronize();

    CU_CHECK(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        std::cout << x[i] << ", ";
    }
    std::cout << "\n";


    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    CU_CHECK(hipFree(d_x));
    free(x);
    return ;
}



void cr_cu(float *a, float *c, float *rhs, int n) {
    int size = n * sizeof(float);
    // Host
    float *x;

    x = (float*)malloc(size);

    // Device
    float *d_a, *d_c, *d_r, *d_x;   // device copies of a, c, rhs
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));
    CU_CHECK(hipMalloc((void **)&d_x, size));

    std::cerr << "CR\n";
    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    cr_ker<<<1, n>>>(d_a, d_c, d_r, d_x, n);

    hipDeviceSynchronize();
    CU_CHECK(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        std::cout << x[i] << ", ";
    }
    std::cout << "\n";

    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    CU_CHECK(hipFree(d_x));
    free(x);
    return ;
}
