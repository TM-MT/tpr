#include "hip/hip_runtime.h"
#include "tpr.cuh"
#include "main.hpp"
#include <iostream>


__global__ void tpr_ker(float *a, float *c, float *rhs, float *x, int n, int s) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int m = n / s;
    int st = idx / s * s;
    int ed = st + s - 1;
    // printf("%d: %d, %d\n", idx, st, ed);

    float tmp_aa, tmp_cc, tmp_rr;
    float inter_ast, inter_cst, inter_rhsst; // bkup
    float inter_aed, inter_ced, inter_rhsed; // bkup

    // stage 1
    if (idx < n) {
        for (int p = 1; p <= static_cast<int>(log2f(static_cast<double>(s))); p++) {
            // reduction
            int u = 1 << (p - 1); // offset
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < st) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = a[lidx];
                ckl = c[lidx];
                rkl = rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx > ed) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = a[ridx];
                ckr = c[ridx];
                rkr = rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * a[idx] - akr * c[idx]);

            tmp_aa = - inv_diag_k * akl* a[idx];
            tmp_cc = - inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);

            __syncthreads();

            // copy back
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;

            __syncthreads();
        }

        // Update E_{st} by E_{ed}
        if (idx == st) {
            int k = st, kr = ed;
            float ak = a[k], akr = a[kr];
            float ck = c[k], ckr = c[kr];
            float rhsk = rhs[k], rhskr = rhs[kr];

            float inv_diag_k = 1.0 / (1.0 - akr * ck);

            tmp_aa = inv_diag_k * ak;
            tmp_cc = -inv_diag_k * ckr * ck;
            tmp_rr = inv_diag_k * (rhsk - rhskr * ck);

            inter_ast = a[st];
            inter_cst = c[st];
            inter_rhsst = rhs[st];

            __syncthreads();

            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;
        }
    }


    // Update E_{st-1} by E_{st}
    if (idx < n - 1 && idx == ed) {
        int k = idx, kr = idx+1; // (k, kr) = (st-1, st)
        float ak = a[k], akr = a[kr];
        float ck = c[k], ckr = c[kr];
        float rhsk = rhs[k], rhskr = rhs[kr];
        float inv_diag_k = 1.0 / (1.0 - akr * ck);

        inter_aed = a[idx];
        inter_ced = c[idx];
        inter_rhsed = rhs[idx];

        a[k] = inv_diag_k * ak;
        c[k] = -inv_diag_k * ckr * ck;
        rhs[k] = inv_diag_k * (rhsk - rhskr * ck);
    }

    // FIX-ME should be block sync
    // __syncthreads();

    // PCR
    if (idx < n && idx == ed) {
        for (int p = static_cast<int>(log2f(static_cast<double>(s))) + 1;
             p <= static_cast<int>(log2f(static_cast<double>(n))); 
             p++) 
        {
            // reduction
            int u = 1 << (p - 1); // offset
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < 0) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = a[lidx];
                ckl = c[lidx];
                rkl = rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx >= n) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = a[ridx];
                ckr = c[ridx];
                rkr = rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * a[idx] - akr * c[idx]);

            tmp_aa = - inv_diag_k * akl* a[idx];
            tmp_cc = - inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);

            // __syncthreads();

            // copy back
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;

            // __syncthreads();
        }
    }
    // __syncthreads();

    // FIX-ME should be block sync

    // stage 3
    cg::thread_block tb = cg::this_thread_block();
    if (idx < n) {
        // copy the answer from stage 2 PCR
        if (idx == ed) {
            x[idx] = rhs[idx];
        }

        // FIX-ME should be block sync
        // __syncthreads();

        if (idx == st) {
            a[idx] = inter_ast;
            c[idx] = inter_cst;
            rhs[idx] = inter_rhsst;
        }

        if (idx == ed && idx != n-1) {
            a[idx] = inter_aed;
            c[idx] = inter_ced;
            rhs[idx] = inter_rhsed;            
        }
    }
    tpr_st3_ker(tb, a, c, rhs, x, n, s);
 
    return ;
}


__device__ void tpr_st3_ker(cg::thread_block tb, float *a, float *c, float *rhs, float *x, int n, int s) {
    int idx = tb.group_index().x * tb.group_dim().x + tb.thread_index().x;
    int st = idx / s * s;
    int ed = st + s - 1;

   if (idx < n) {
        int lidx = max(0, st - 1);

        float key = 1.0 / c[ed] * (rhs[ed] - a[ed] * x[lidx] - x[ed]);
        if (c[ed] == 0.0) {
            key = 0.0;
        }

        x[idx] = rhs[idx] - a[idx] * x[lidx] - c[idx] * key;
    }
    return ;
}


__global__ void pcr_ker(float *a, float *c, float *rhs, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp_aa, tmp_cc, tmp_rr;

    if (idx < n) {
        for (int p = 1; p <= static_cast<int>(log2f(static_cast<double>(n))); p++) {
            // reduction
            int u = 1 << (p - 1); // offset
            int lidx = idx - u;
            float akl, ckl, rkl;
            if (lidx < 0) {
                akl = -1.0;
                ckl = 0.0;
                rkl = 0.0;
            } else {
                akl = a[lidx];
                ckl = c[lidx];
                rkl = rhs[lidx];
            }
            int ridx = idx + u;
            float akr, ckr, rkr;
            if (ridx >= n) {
                akr = 0.0;
                ckr = -1.0;
                rkr = 0.0;
            } else {
                akr = a[ridx];
                ckr = c[ridx];
                rkr = rhs[ridx];
            }

            float inv_diag_k = 1.0 / (1.0 - ckl * a[idx] - akr * c[idx]);

            tmp_aa = - inv_diag_k * akl* a[idx];
            tmp_cc = - inv_diag_k * ckr * c[idx];
            tmp_rr = inv_diag_k * (rhs[idx] - rkl * a[idx] - rkr * c[idx]);

            __syncthreads();

            // copy back
            a[idx] = tmp_aa;
            c[idx] = tmp_cc;
            rhs[idx] = tmp_rr;

            __syncthreads();
        }
    }
}



#define CU_CHECK( expr ) { hipError_t t = expr;\
    if (t != hipSuccess) {\
        fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", hipGetErrorString(t), t, __FILE__, __LINE__); \
        exit(EXIT_FAILURE);\
    } \
}


int main() {
    int n = 1024;
    struct TRIDIAG_SYSTEM *sys = (struct TRIDIAG_SYSTEM *)malloc(sizeof(struct TRIDIAG_SYSTEM));
    setup(sys, n);
    for (int s = 128; s <= n; s *= 2) {
        assign(sys);
        tpr_cu(sys->a, sys->c, sys->rhs, n, s);
    }

    assign(sys);
    pcr_cu(sys->a, sys->c, sys->rhs, n);

    clean(sys);
    free(sys);

}


int setup(struct TRIDIAG_SYSTEM *sys, int n) {
    sys->a = (real *)malloc(n * sizeof(real));
    sys->diag = (real *)malloc(n * sizeof(real));
    sys->c = (real *)malloc(n * sizeof(real));
    sys->rhs = (real *)malloc(n * sizeof(real));
    sys->n = n;

    return sys_null_check(sys);
}

int assign(struct TRIDIAG_SYSTEM *sys) {
    int n = sys->n;
    for (int i = 0; i < n; i++) {
        sys->a[i] = -1.0/6.0;
        sys->c[i] = -1.0/6.0;
        sys->diag[i] = 1.0;
        sys->rhs[i] = 1.0 * (i+1);
    }
    sys->a[0] = 0.0;
    sys->c[n-1] = 0.0;

    return 0;
}



int clean(struct TRIDIAG_SYSTEM *sys) {
    for (auto p: { sys->a, sys->diag, sys->c, sys->rhs }) {
        free(p);
    }

    sys->a = nullptr;
    sys->diag = nullptr;
    sys->c = nullptr;
    sys->rhs = nullptr;

    return 0;
}


bool sys_null_check(struct TRIDIAG_SYSTEM *sys) {
    for (auto p: { sys->a, sys->diag, sys->c, sys->rhs }) {
        if (p == nullptr) {
            return false;
        }
    }
    return true;
}



void tpr_cu(float *a, float *c, float *rhs, int n, int s) {
    int size = n * sizeof(float);
    // Host
    float *x;

    x = (float*)malloc(size);

    // Device
    float *d_a, *d_c, *d_r;   // device copies of a, c, rhs
    float *d_x;
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));
    CU_CHECK(hipMalloc((void **)&d_x, size));

    std::cerr << "TPR: s=" << s << "\n";
    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice)); 
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice)); 

    hipDeviceSynchronize();

    // launch
    tpr_ker<<<n / s, s>>>(d_a, d_c, d_r, d_x, n, s);

    hipDeviceSynchronize();

    CU_CHECK(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        std::cout << x[i] << ", ";
    }
    std::cout << "\n";


    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    CU_CHECK(hipFree(d_x));
    free(x);
    return ;
}



void pcr_cu(float *a, float *c, float *rhs, int n) {
    int size = n * sizeof(float);
    // Host
    float *x;

    x = (float*)malloc(size);

    // Device
    float *d_a, *d_c, *d_r;   // device copies of a, c, rhs
    CU_CHECK(hipMalloc((void **)&d_a, size));
    CU_CHECK(hipMalloc((void **)&d_c, size));
    CU_CHECK(hipMalloc((void **)&d_r, size));

    std::cerr << "PCR\n";
    CU_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));
    CU_CHECK(hipMemcpy(d_r, rhs, size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    pcr_ker<<<1, n>>>(d_a, d_c, d_r, n);

    hipDeviceSynchronize();
    CU_CHECK(hipMemcpy(x, d_r, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        std::cout << x[i] << ", ";
    }
    std::cout << "\n";

    CU_CHECK(hipFree(d_a));
    CU_CHECK(hipFree(d_c));
    CU_CHECK(hipFree(d_r));
    free(x);
    return ;
}
