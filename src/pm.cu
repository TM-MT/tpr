#include "pm.cuh"
#include "pm.hpp"

#define CU_CHECK(expr)                                                     \
    {                                                                      \
        hipError_t t = expr;                                              \
        if (t != hipSuccess) {                                            \
            fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", \
                    hipGetErrorString(t), t, __FILE__, __LINE__);         \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

namespace pmcpp {
class DeviceTimer {
    hipEvent_t ev_start, ev_stop;

   public:
    DeviceTimer() {
        CU_CHECK(hipEventCreate(&ev_start));
        CU_CHECK(hipEventCreate(&ev_stop));
    }

    ~DeviceTimer() {
        CU_CHECK(hipEventDestroy(ev_start));
        CU_CHECK(hipEventDestroy(ev_stop));
    }

    void start() { CU_CHECK(hipEventRecord(ev_start, hipEventDefault)); }

    void stop() {
        CU_CHECK(hipEventRecord(ev_stop, hipEventDefault));
        hipEventSynchronize(ev_stop);
    }

    void get_elapsed_time(time_ms &elapsed) {
        hipEventElapsedTime(&elapsed, ev_start, ev_stop);
    }

    void stop_and_elapsed(time_ms &elapsed) {
        stop();
        get_elapsed_time(elapsed);
    }
};
}  // namespace pmcpp

#undef CU_CHECK
